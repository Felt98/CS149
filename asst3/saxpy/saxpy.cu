#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include  <driver_functions.h>
#include "cuda_debug.h"
#include "CycleTimer.h"

// #define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
// inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//     printf("debug----------------------\n");
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr, "CUDA Error: %s at %s:%d\n", 
//         hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

const int thread_num=1024;
int N = 100 * 1000 * 1000;
// return GB/sec
float GBPerSec(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("index: %d \n",index);

    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    if (index < N)
       result[index] = alpha * x[index] + y[index];
}


// saxpyCuda --
//
// This function is regular C code running on the CPU.  It allocates
// memory on the GPU using CUDA API functions, uses CUDA API functions
// to transfer data from the CPU's memory address space to GPU memory
// address space, and launches the CUDA kernel function on the GPU.
void saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    // must read both input arrays (xarray and yarray) and write to
    // output array (resultarray)
    int totalBytes = sizeof(float) * 3 * N;
    int arryBytes = sizeof(float)*N;

    // compute number of blocks and threads per block.  In this
    // application we've hardcoded thread blocks to contain 512 CUDA
    // threads.
    const int threadsPerBlock = 512;

    // Notice the round up here.  The code needs to compute the number
    // of threads blocks needed such that there is one thread per
    // element of the arrays.  This code is written to work for values
    // of N that are not multiples of threadPerBlock.
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // These are pointers that will be pointers to memory allocated
    // *one the GPU*.  You should allocate these pointers via
    // hipMalloc.  You can access the resulting buffers from CUDA
    // device kernel code (see the kernel function saxpy_kernel()
    // above) but you cannot access the contents these buffers from
    // this thread. CPU threads cannot issue loads and stores from GPU
    // memory!
    float* device_x = nullptr;
    float* device_y = nullptr;
    float* device_result = nullptr;


    // CS149 TODO: allocate device memory buffers on the GPU using hipMalloc.
    //
    // We highly recommend taking a look at NVIDIA's
    // tutorial, which clearly walks you through the few lines of code
    // you need to write for this part of the assignment:
    //
    // https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
    //
        
    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();
    hipMalloc(&device_x,arryBytes);

    hipMalloc(&device_y,arryBytes);

    hipMalloc(&device_result,arryBytes);

    hipMemcpy(device_x,xarray,arryBytes,hipMemcpyHostToDevice);
    
    hipMemcpy(device_y,yarray,arryBytes,hipMemcpyHostToDevice);
    

    //
    // CS149 TODO: copy input arrays to the GPU using hipMemcpy
    //

   
    // run CUDA kernel. (notice the <<< >>> brackets indicating a CUDA
    // kernel launch) Execution on the GPU occurs here.
    double startTime2 = CycleTimer::currentSeconds();
    
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double endTime2 = CycleTimer::currentSeconds();

    double overallDuration2 = endTime2 - startTime2;
    printf("Effective compute time by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration2, GBPerSec(totalBytes, overallDuration2));

    //
    // CS149 TODO: copy result from GPU back to CPU using hipMemcpy
    //

    
    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();
    //hipMemcpy(resultarray,device_result,arryBytes,hipMemcpyDeviceToHost);
    cudaCheckError(hipMemcpy(resultarray,device_result,arryBytes,hipMemcpyHostToDevice));
    // hipError_t errCode = hipPeekAtLastError();
    // if (errCode != hipSuccess) {
    //     fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
	// 	errCode, hipGetErrorString(errCode));
    // }

    double overallDuration = endTime - startTime;
    printf("Effective BW by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration));

    //
    // CS149 TODO: free memory buffers on the GPU using hipFree
    //
    cudaCheckError(hipFree(device_x));
    cudaCheckError(hipFree(device_x));
    hipFree(device_y);
    hipFree(device_result);
    
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
